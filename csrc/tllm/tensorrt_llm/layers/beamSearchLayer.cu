#include "hip/hip_runtime.h"
/*
 * Copyright (c) 2019-2024, NVIDIA CORPORATION.  All rights reserved.
 *
 * Licensed under the Apache License, Version 2.0 (the "License");
 * you may not use this file except in compliance with the License.
 * You may obtain a copy of the License at
 *
 *     http://www.apache.org/licenses/LICENSE-2.0
 *
 * Unless required by applicable law or agreed to in writing, software
 * distributed under the License is distributed on an "AS IS" BASIS,
 * WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
 * See the License for the specific language governing permissions and
 * limitations under the License.
 */

#include "tensorrt_llm/kernels/beamSearchKernels.h"
#include "tensorrt_llm/layers/beamSearchLayer.h"
#include "tensorrt_llm/layers/defaultDecodingParams.h"
#include "tensorrt_llm/layers/layerUtils.h"
#include <limits>

using namespace tensorrt_llm::common;
using namespace tensorrt_llm::kernels;

namespace tensorrt_llm::layers
{

template <typename T>
BeamSearchLayer<T>::BeamSearchLayer(
    DecoderDomain const& decoderDomain, hipStream_t stream, std::shared_ptr<IAllocator> allocator)
    : BaseLayer(decoderDomain, stream, std::move(allocator))
    , mVocabSize(decoderDomain.getVocabSize())
    , mVocabSizePadded(decoderDomain.getVocabSizePadded())
{
    TLLM_LOG_TRACE(__PRETTY_FUNCTION__);

    mDiversityRateHost.resize(mDecoderDomain.getBatchSize());
    mLengthPenaltyHost.resize(mDecoderDomain.getBatchSize());
    mEarlyStoppingHost.resize(mDecoderDomain.getBatchSize());
    allocateBuffer(mDecoderDomain.getBatchSize(), mDecoderDomain.getBeamWidth());

    TLLM_CHECK_WITH_INFO(mDecoderDomain.getBeamWidth() <= nMaxBeamWidth,
        std::string("Beam width is larger than the maximum supported (" + std::to_string(mDecoderDomain.getBeamWidth())
            + " > " + std::to_string(nMaxBeamWidth) + ")."));
}

template <typename T>
BeamSearchLayer<T>::~BeamSearchLayer()
{
    TLLM_LOG_TRACE(__PRETTY_FUNCTION__);
}

template <typename T>
void BeamSearchLayer<T>::setup(runtime::SizeType32 const batchSize, runtime::SizeType32 const beamWidth,
    runtime::SizeType32 const* batchSlots, std::shared_ptr<BaseSetupParams> const& baseSetupParams)
{
    TLLM_LOG_TRACE("%s start", __PRETTY_FUNCTION__);
    TLLM_CHECK_WITH_INFO(beamWidth <= mDecoderDomain.getBeamWidth(),
        std::string("Beam width is larger than the constructed for (" + std::to_string(beamWidth) + " > "
            + std::to_string(mDecoderDomain.getBeamWidth()) + ")."));

    auto setupParams = std::dynamic_pointer_cast<BeamSearchSetupParams>(baseSetupParams);

    auto constexpr fltMax = std::numeric_limits<float>::max();
    auto constexpr fltMin = std::numeric_limits<float>::lowest();
    auto constexpr fltEpsilon = std::numeric_limits<float>::epsilon();

    std::vector<SizeType32> batchSlotsVec(batchSize);
    std::iota(batchSlotsVec.begin(), batchSlotsVec.end(), 0);
    auto batchSlotsHost = batchSlots ? batchSlots : batchSlotsVec.data();

    FillBuffers const fillBuffers{batchSize, mDecoderDomain.getBatchSize(), mStream};
    fillBuffers(setupParams->beamSearchDiversityRate, DefaultDecodingParams::getBeamSearchDiversity(),
        mDiversityRateHost, mDiversityRateDevice, batchSlotsHost, std::make_pair(-fltEpsilon, fltMax),
        "diversity rate");
    fillBuffers(setupParams->lengthPenalty, DefaultDecodingParams::getLengthPenalty(), mLengthPenaltyHost,
        mLengthPenaltyDevice, batchSlotsHost, std::make_pair(fltMin, fltMax), "length penalty");
    fillBuffers(setupParams->earlyStopping, DefaultDecodingParams::getEarlyStopping(), mEarlyStoppingHost,
        mEarlyStoppingDevice, batchSlotsHost, std::make_pair(0, std::numeric_limits<int>::max()), "early stopping");

    TLLM_LOG_TRACE("%s stop", __PRETTY_FUNCTION__);
}

__global__ void updateCacheIndirectionKernel(
    int* tgtCI, int const* srcCI, BeamHypotheses bh, int const nMaxAttentionWindow, int const nSinkTokenLength)
{
    // Update indirections from steps `bh.inputLength[indexBatchBeam]` to step `sequenceLengths[indexBatchBeam]`
    int const step = threadIdx.x + blockIdx.x * blockDim.x;
    int const nBM{bh.nBeamWidth};
    int const nMSL{bh.nMaxSeqLen};
    int const indexBatch = blockIdx.y;
    int const batchSlot = bh.batchSlots ? bh.batchSlots[indexBatch] : indexBatch;
    int const indexBeam = blockIdx.z;
    int const indexBatchBeam = batchSlot * nBM + indexBeam;
    int const lastStep{bh.sequenceLengths[indexBatchBeam] - 1}; // the sequenceLengths is updated, need to minus 1

    // Return early when the indexBatchBeam or step is out of the bound
    // No update for the indices of context part since KV Cache is shared
    if (step >= nMSL || step < bh.inputLengths[indexBatchBeam] || step < (nMSL - nMaxAttentionWindow)
        || bh.finished[indexBatchBeam].isFinished())
    {
        return;
    }

    // Keep all past tokens by parentIdsPtr
    int const indexBeamSrc = bh.parentIdsPtr[batchSlot][indexBeam * nMSL + lastStep];
    int const stepCirc = (step >= nSinkTokenLength)
        ? nSinkTokenLength + (step - nSinkTokenLength) % (nMaxAttentionWindow - nSinkTokenLength)
        : step;
    // Consider cyclic kv cache for the indir tables
    uint32_t const tgtOffset = batchSlot * nBM * nMaxAttentionWindow + indexBeam * nMaxAttentionWindow + stepCirc;
    uint32_t const srcOffset = batchSlot * nBM * nMaxAttentionWindow + indexBeamSrc * nMaxAttentionWindow + stepCirc;
    tgtCI[tgtOffset] = (step == lastStep) ? indexBeam : srcCI[srcOffset];
}

template <typename T>
void BeamSearchLayer<T>::forwardAsync(
    std::shared_ptr<BaseDecodingOutputs> const& baseOutputs, std::shared_ptr<BaseDecodingInputs> const& baseInputs)
{
    TLLM_LOG_TRACE("%s start", __PRETTY_FUNCTION__);

    auto ip = std::dynamic_pointer_cast<DecodingInputs>(baseInputs);
    auto op = std::dynamic_pointer_cast<BeamSearchOutputs>(baseOutputs);
    auto const localDecoderDomain = getLocalDecoderDomain(ip, mDecoderDomain);

    TLLM_CHECK_WITH_INFO(localDecoderDomain.getBeamWidth() > 1,
        "Decoding mode is beam search, but beamWidth <= 1 (%d <= 1)", localDecoderDomain.getBeamWidth());
    TLLM_CHECK_WITH_INFO(ip->srcCacheIndirection.has_value(), "srcCacheIndirection is mandatory in beam search.");
    TLLM_CHECK_WITH_INFO(op->parentIds.has_value(), "parentIds tensor is mandatory in beam search.");
    TLLM_CHECK_WITH_INFO(op->finished.has_value(), "finished tensor is mandatory in beam search.");
    TLLM_CHECK_WITH_INFO(op->cumLogProbs.has_value(), "cumLogProbs tensor is mandatory in beam search.");
    TLLM_CHECK_WITH_INFO(op->beamHypotheses, std::string("Output BeamHypotheses is not set."));
    TLLM_CHECK_WITH_INFO(op->sequenceLength->template getPtr<int>() != nullptr || mLengthPenaltyDevice == nullptr,
        std::string("Current sequence lengths must be set for length penalty computation."));
    TLLM_CHECK_WITH_INFO(ip->ite == 0, "Pipeline Parallelism is not supported yet !");

    BeamHypotheses bh;
    // bh's members not used in function: outputIds, logProbs, outputIdsUnfinish, parentIdsUnfinish
    bh.outputIdsCBA = op->beamHypotheses->outputIdsCBA;
    bh.logProbsCBA = op->beamHypotheses->logProbsCBA;
    bh.sequenceLengthsCBA = op->beamHypotheses->sequenceLengthsCBA;
    bh.cumLogProbsCBA = op->beamHypotheses->cumLogProbsCBA;
    bh.normedScoresCBA = op->beamHypotheses->normedScoresCBA;
    bh.numBeamsCBA = op->beamHypotheses->numBeamsCBA;
    bh.minNormedScoresCBA = op->beamHypotheses->minNormedScoresCBA;
    bh.batchDones = op->beamHypotheses->batchDones;
    bh.nMaxBatchSize = static_cast<std::int32_t>(op->outputIdsPtr.shape[0]);
    bh.nBatchSize = ip->localBatchSize;
    bh.batchSlots = ip->batchSlots ? ip->batchSlots->template getPtr<SizeType32 const>() : nullptr;
    bh.nBeamWidth = static_cast<std::int32_t>(op->outputIds.shape[1]);
    bh.nMaxSeqLen = static_cast<std::int32_t>(op->outputIds.shape[2]);
    bh.nVocabSize = mVocabSizePadded;
    bh.diversityRates = mDiversityRateDevice;
    bh.lengthPenalties = mLengthPenaltyDevice;
    bh.earlyStoppings = mEarlyStoppingDevice;
    bh.inputLengths = ip->inputLengths->template getPtr<int const>();
    bh.endIds = ip->endIds.template getPtr<int const>();
    bh.logProbsTiled = (op->outputLogProbsTiled) ? op->outputLogProbsTiled->template getPtr<float>() : nullptr;
    bh.sequenceLengths = op->sequenceLength->template getPtr<int>();
    bh.cumLogProbs = op->cumLogProbs->template getPtr<float>();
    bh.finished = reinterpret_cast<FinishedState*>(op->finished->template getPtr<FinishedState::UnderlyingType>());
    bh.outputIdsPtr = op->outputIdsPtr.template getPtr<int*>();
    bh.parentIdsPtr = op->parentIdsPtr.template getPtr<int*>();

    T const* logits = ip->logits->template getPtr<T>();
    T const* bias = static_cast<T const*>(nullptr);
    TLLM_CHECK_WITH_INFO(mWorkspaceSize >= 2 * bh.nBatchSize * bh.nBeamWidth * bh.nBeamWidth * 2,
        fmtstr("Workspace size (%lu) is not enough for topk softmax required (%lu).", (uint64_t) mWorkspaceSize,
            (uint64_t) (2 * bh.nMaxBatchSize * bh.nBeamWidth * bh.nBeamWidth * 2)));

    invokeTopkSoftMax(logits, bias, mWorkspace, bh, mStream);
    sync_check_cuda_error();

    if (bh.nBeamWidth > 1)
    {
        auto tgtCI = op->tgtCacheIndirection.template getPtr<int>();
        auto srcCI = ip->srcCacheIndirection->template getPtr<int const>();
        dim3 const grid(roundUp(bh.nMaxSeqLen, 32), bh.nBatchSize, bh.nBeamWidth);
        updateCacheIndirectionKernel<<<grid, 32, 0, mStream>>>(
            tgtCI, srcCI, bh, ip->maxAttentionWindow, ip->sinkTokenLength);
        sync_check_cuda_error();
    }

    TLLM_LOG_TRACE("%s stop", __PRETTY_FUNCTION__);
}

template <typename T>
void BeamSearchLayer<T>::allocateBuffer(runtime::SizeType32 const batchSize, runtime::SizeType32 const beamWidth)
{
    TLLM_LOG_TRACE("%s start", __PRETTY_FUNCTION__);
    int const nPadBeamWidth = padToNextPowerOfTwo(beamWidth);
    // Unit of mWorkspaceSize is number of elements (not Byte), align to 4 for further optimization
    size_t nTopK = batchSize * nPadBeamWidth * nPadBeamWidth * 2;
    size_t nTempBuffer = batchSize * nPadBeamWidth * nMaxVocabPartForStage1FastKernel * (2 * (nPadBeamWidth * 2) + 2);
    mWorkspaceSize = roundUp(nTopK, 4) * 2 + roundUp(nTempBuffer, 4);
    mWorkspace = mAllocator->reMalloc(mWorkspace, sizeof(float) * mWorkspaceSize, true);
    mDiversityRateDevice
        = mAllocator->reMalloc(mDiversityRateDevice, sizeof(float) * mDecoderDomain.getBatchSize(), false);
    mLengthPenaltyDevice
        = mAllocator->reMalloc(mLengthPenaltyDevice, sizeof(float) * mDecoderDomain.getBatchSize(), false);
    mEarlyStoppingDevice
        = mAllocator->reMalloc(mEarlyStoppingDevice, sizeof(int) * mDecoderDomain.getBatchSize(), false);
    mIsAllocateBuffer = true;
    TLLM_LOG_TRACE("%s stop", __PRETTY_FUNCTION__);
}

template <typename T>
void BeamSearchLayer<T>::freeBuffer()
{
    TLLM_LOG_TRACE("%s start", __PRETTY_FUNCTION__);
    if (mIsAllocateBuffer)
    {
        mAllocator->free((void**) (&mWorkspace));
        mAllocator->free((void**) (&mDiversityRateDevice));
        mAllocator->free((void**) (&mLengthPenaltyDevice));
        mAllocator->free((void**) (&mEarlyStoppingDevice));
        mIsAllocateBuffer = false;
    }
    TLLM_LOG_TRACE("%s stop", __PRETTY_FUNCTION__);
}

template class BeamSearchLayer<float>;
template class BeamSearchLayer<half>;

} // namespace tensorrt_llm::layers
